#include "hip/hip_runtime.h"
#ifdef __CUDA_ARCH__
#define HOST_DEVICE __host__ __device__
#else
#define HOST_DEVICE
#endif

#include "Parallel.h"
#include "Docking.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Kernel de ejemplo: cada hilo asigna un score dummy
__global__ void dockingKernel(const int numProteins, const int numLigands, float* scores) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = numProteins * numLigands;
    if (idx < total) {
        // Cálculo dummy: en una implementación real se usarían los datos de las moléculas.
        scores[idx] = 42.0f;
    }
}

void cudaDocking(const std::vector<Molecule>& proteins,
                 const std::vector<Molecule>& ligands,
                 std::vector<float>& scores) {
    std::cout << "Ejecutando docking con CUDA..." << std::endl;
    size_t total = proteins.size() * ligands.size();
    scores.resize(total);

    float* d_scores = nullptr;
    hipMalloc((void**)&d_scores, total * sizeof(float));

    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;
    dockingKernel<<<gridSize, blockSize>>>(proteins.size(), ligands.size(), d_scores);
    hipDeviceSynchronize();

    hipMemcpy(scores.data(), d_scores, total * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_scores);
}