#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <cstdlib>
#include <cassert>
#include <hip/hip_runtime.h>

// Se incluyen las implementaciones existentes para manejo de datos y moléculas.
// Se asume que DataManager, Molecule, Atom, Docking y Utils se han implementado en CPU.
#include "DataManager.h"   // Funciones de carga de proteínas y ligandos
#include "Molecule.h"      // Definición de Molecule y Atom (struct con x, y, z y element)
#include "Docking.h"       // Versión secuencial (opcional para comparar)
#include "Utils.h"         // parseArguments, Timer, analyzeDockingResults, etc.

using namespace std;

//-----------------------------------------------------------------------------
// Estructura que usaremos en device para almacenar las coordenadas de un átomo.
// Se omite información adicional (por ejemplo, el elemento) que no se usa en el cálculo.
struct AtomGPU {
    float x, y, z;
};

//-----------------------------------------------------------------------------
// Kernel CUDA que ejecuta el docking entre cada proteína y ligando, sabiendo que
// cada molécula tiene un número constante de átomos.
// Los parámetros atomsPerProtein y atomsPerLigand son valores conocidos.
__global__
void dockingKernelConstant(const AtomGPU* proteinAtoms,
                           const AtomGPU* ligandAtoms,
                           float* scores,
                           int numProteins, int numLigands,
                           int atomsPerProtein, int atomsPerLigand)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalDockings = numProteins * numLigands;
    if (tid >= totalDockings)
        return;
    
    // A partir de tid se determina la proteína y el ligando correspondientes
    int proteinIdx = tid / numLigands;
    int ligandIdx  = tid % numLigands;
    
    float energy = 0.0f;
    const float epsilon = 1.0f;
    
    // Calcular los offsets en los arreglos aplanados, pues cada molécula es contigua
    int proteinOffset = proteinIdx * atomsPerProtein;
    int ligandOffset  = ligandIdx  * atomsPerLigand;
    
    // Cálculo del potencial de Lennard-Jones para cada par de átomos de la pareja (ligando, proteína)
    for (int i = 0; i < atomsPerLigand; i++) {
        AtomGPU aL = ligandAtoms[ligandOffset + i];
        for (int j = 0; j < atomsPerProtein; j++) {
            AtomGPU aP = proteinAtoms[proteinOffset + j];
            float dx = aL.x - aP.x;
            float dy = aL.y - aP.y;
            float dz = aL.z - aP.z;
            float r2 = dx * dx + dy * dy + dz * dz;
            if (r2 < 1e-6f)
                continue;
            float r6  = r2 * r2 * r2;
            float r12 = r6 * r6;
            float ljEnergy = 4.0f * epsilon * ((1.0f / r12) - (1.0f / r6));
            energy += ljEnergy;
        }
    }
    
    scores[tid] = energy;
}

//-----------------------------------------------------------------------------
// Función principal donde se asume que el número de átomos por molécula es constante.
// Se define, por ejemplo, ATOMS_PER_PROTEIN = 2000 y ATOMS_PER_LIGAND = 30.
int main(int argc, char* argv[])
{
    // Definición de constantes para el número de átomos
    const int ATOMS_PER_PROTEIN = 1000; 
    const int ATOMS_PER_LIGAND  = 100;
    
    // Variables de entrada (directorios, modo verbose, etc.)
    string proteinsDir, ligandsDir;
    bool verbose = false;
    
    parseArguments(argc, argv, proteinsDir, ligandsDir, verbose);
    
    // Carga de moléculas usando DataManager (implementado en CPU)
    DataManager dataManager;
    vector<Molecule> proteins;
    vector<Molecule> ligands;
    
    if (!dataManager.loadProteins(proteinsDir, proteins)) {
        cerr << "Error loading proteins." << endl;
        exit(EXIT_FAILURE);
    }
    if (!dataManager.loadLigands(ligandsDir, ligands)) {
        cerr << "Error loading ligands." << endl;
        exit(EXIT_FAILURE);
    }
    
    int numProteins = proteins.size();
    int numLigands  = ligands.size();
    int totalDockings = numProteins * numLigands;
    
    // Se valida que cada proteína y ligando tenga el número de átomos esperado:
    for (const auto &protein : proteins) {
         assert(protein.getAtoms().size() == ATOMS_PER_PROTEIN);
    }
    for (const auto &ligand : ligands) {
         assert(ligand.getAtoms().size() == ATOMS_PER_LIGAND);
    }
    
    // Se construyen arreglos aplanados de átomos para proteínas y ligandos.
    // Cada proteína ocupará un bloque de ATOMS_PER_PROTEIN y similarmente para ligandos.
    vector<AtomGPU> flatProteinAtoms(numProteins * ATOMS_PER_PROTEIN);
    vector<AtomGPU> flatLigandAtoms(numLigands * ATOMS_PER_LIGAND);
    
    // Transferir datos de proteínas a un arreglo lineal
    for (int i = 0; i < numProteins; i++) {
         const vector<Atom>& atoms = proteins[i].getAtoms();
         for (int j = 0; j < ATOMS_PER_PROTEIN; j++) {
             flatProteinAtoms[i * ATOMS_PER_PROTEIN + j].x = atoms[j].x;
             flatProteinAtoms[i * ATOMS_PER_PROTEIN + j].y = atoms[j].y;
             flatProteinAtoms[i * ATOMS_PER_PROTEIN + j].z = atoms[j].z;
         }
    }
    
    // Transferir datos de ligandos a un arreglo lineal
    for (int i = 0; i < numLigands; i++) {
         const vector<Atom>& atoms = ligands[i].getAtoms();
         for (int j = 0; j < ATOMS_PER_LIGAND; j++) {
             flatLigandAtoms[i * ATOMS_PER_LIGAND + j].x = atoms[j].x;
             flatLigandAtoms[i * ATOMS_PER_LIGAND + j].y = atoms[j].y;
             flatLigandAtoms[i * ATOMS_PER_LIGAND + j].z = atoms[j].z;
         }
    }
    
    // Reservar memoria en la GPU para los átomos aplanados y para el vector de scores
    AtomGPU *d_proteinAtoms, *d_ligandAtoms;
    float* d_scores;
    size_t sizeProteinAtoms = flatProteinAtoms.size() * sizeof(AtomGPU);
    size_t sizeLigandAtoms  = flatLigandAtoms.size() * sizeof(AtomGPU);
    size_t sizeScores       = totalDockings * sizeof(float);
    
    hipError_t err;
    err = hipMalloc((void**)&d_proteinAtoms, sizeProteinAtoms);
    if (err != hipSuccess) {
         cerr << "Error allocando memoria para d_proteinAtoms: " << hipGetErrorString(err) << endl;
         exit(EXIT_FAILURE);
    }
    err = hipMalloc((void**)&d_ligandAtoms, sizeLigandAtoms);
    if (err != hipSuccess) {
         cerr << "Error allocando memoria para d_ligandAtoms: " << hipGetErrorString(err) << endl;
         exit(EXIT_FAILURE);
    }
    err = hipMalloc((void**)&d_scores, sizeScores);
    if (err != hipSuccess) {
         cerr << "Error allocando memoria para d_scores: " << hipGetErrorString(err) << endl;
         exit(EXIT_FAILURE);
    }
    
    // Transferir datos desde host a device
    hipMemcpy(d_proteinAtoms, flatProteinAtoms.data(), sizeProteinAtoms, hipMemcpyHostToDevice);
    hipMemcpy(d_ligandAtoms, flatLigandAtoms.data(), sizeLigandAtoms, hipMemcpyHostToDevice);
    
    // Configurar los parámetros para el lanzamiento del kernel: cada hilo calcula una combinación proteína-ligando
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalDockings + threadsPerBlock - 1) / threadsPerBlock;
    
    cout << "Modo CUDA con átomos constantes por molécula" << endl;
    
    // Medición del tiempo de ejecución en la GPU usando eventos
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    dockingKernelConstant<<<blocksPerGrid, threadsPerBlock>>>(d_proteinAtoms,
                                                              d_ligandAtoms,
                                                              d_scores,
                                                              numProteins,
                                                              numLigands,
                                                              ATOMS_PER_PROTEIN,
                                                              ATOMS_PER_LIGAND);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Tiempo de ejecución (kernel GPU): " << milliseconds << " ms" << endl;
    
    // Copiar resultados (scores) desde la GPU al host
    vector<float> scores(totalDockings);
    hipMemcpy(scores.data(), d_scores, sizeScores, hipMemcpyDeviceToHost);
    
    // (Opcional) Análisis de resultados
    if (verbose)
         analyzeDockingResults(scores, numProteins, numLigands);
    
    // Liberar memoria en la GPU y destruir eventos
    hipFree(d_proteinAtoms);
    hipFree(d_ligandAtoms);
    hipFree(d_scores);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
